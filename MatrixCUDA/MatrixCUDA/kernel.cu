#include "hip/hip_runtime.h"
﻿// This program computes a simple version of matrix multiplication
// By: Nick from CoffeeBeforeArch

#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

__global__ void matrixMul(const int* a, const int* b, int* c, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    
    c[row * N + col] = 0;
    for (int k = 0; k < N; k++) {
        
        c[row * N + col] += a[row * N + k] * b[k * N + col];
    }
}

// Проверка результатов на CPU
void verify_result(vector<int>& a, vector<int>& b, vector<int>& c, int N) {
    
    for (int i = 0; i < N; i++) {
        
        for (int j = 0; j < N; j++) {
            
            int tmp = 0;
            for (int k = 0; k < N; k++) {
                
                tmp += a[i * N + k] * b[k * N + j];
            }

            
            assert(tmp == c[i * N + j]);
            //cout << tmp << " ";
        }
    }
}

int main() {
    for (int t = 0; t < 5; t++) {
        
        int N = 1 << 10;

        
        size_t bytes = N * N * sizeof(int);

        
        vector<int> h_a(N * N);
        vector<int> h_b(N * N);
        vector<int> h_c(N * N);

        
        generate(h_a.begin(), h_a.end(), []() { return rand() % 100; });
        generate(h_b.begin(), h_b.end(), []() { return rand() % 100; });

        
        int* d_a, * d_b, * d_c;
        hipMalloc(&d_a, bytes);
        hipMalloc(&d_b, bytes);
        hipMalloc(&d_c, bytes);

        
        hipMemcpy(d_a, h_a.data(), bytes, hipMemcpyHostToDevice);
        hipMemcpy(d_b, h_b.data(), bytes, hipMemcpyHostToDevice);

        
        int THREADS = 32;

        
        int BLOCKS = N / THREADS;

        
        dim3 threads(THREADS, THREADS);
        dim3 blocks(BLOCKS, BLOCKS);

        
        matrixMul << <blocks, threads >> > (d_a, d_b, d_c, N);

        
        hipMemcpy(h_c.data(), d_c, bytes, hipMemcpyDeviceToHost);

        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
    }
    return 0;
}